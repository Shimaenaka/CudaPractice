
#include <hip/hip_runtime.h>
// #include "graph.h"
// #include <cuda_runtime.h>

// void allocateGraphOnGPU(Graph &graph, int **d_edges, int **d_offsets) {
//     cudaMalloc((void **)d_edges, graph.edges.size() * sizeof(int));
//     cudaMalloc((void **)d_offsets, graph.edgeOffsets.size() * sizeof(int));

//     cudaMemcpy(*d_edges, graph.edges.data(), graph.edges.size() * sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(*d_offsets, graph.edgeOffsets.data(), graph.edgeOffsets.size() * sizeof(int), cudaMemcpyHostToDevice);
// }

// void freeGraphOnGPU(int *d_edges, int *d_offsets) {
//     cudaFree(d_edges);
//     cudaFree(d_offsets);
// }