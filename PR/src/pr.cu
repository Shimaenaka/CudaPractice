#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "graph.h"

void allocateGraphOnGPU(Graph &graph, int **d_edges, int **d_offsets) {
    hipMalloc((void **)d_edges, graph.edges.size() * sizeof(int));
    hipMalloc((void **)d_offsets, graph.edgeOffsets.size() * sizeof(int));

    hipMemcpy(*d_edges, graph.edges.data(), graph.edges.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(*d_offsets, graph.edgeOffsets.data(), graph.edgeOffsets.size() * sizeof(int), hipMemcpyHostToDevice);
}

void freeGraphOnGPU(int *d_edges, int *d_offsets) {
    hipFree(d_edges);
    hipFree(d_offsets);
}

__global__ void pageRankKernel(int numNodes, int *d_edges, int *d_offsets, float *d_pr, float *d_prNext, float damping, float equalProb) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node < numNodes) {
        float sum = 0.0f;
        int start = d_offsets[node];
        int end = d_offsets[node + 1];
        for (int edge = start; edge < end; ++edge) {
            int neighbor = d_edges[edge];
            sum += d_pr[neighbor] / (d_offsets[neighbor + 1] - d_offsets[neighbor]);
        }
        d_prNext[node] = damping * sum + (1.0f - damping) * equalProb;
    }
}

void pageRank(Graph &graph, int iterations = 100, float damping = 0.85) {
    int *d_edges, *d_offsets;
    allocateGraphOnGPU(graph, &d_edges, &d_offsets);

    float *d_pr, *d_prNext;
    hipMalloc((void **)&d_pr, graph.numNodes * sizeof(float));
    hipMalloc((void **)&d_prNext, graph.numNodes * sizeof(float));

    std::vector<float> pr(graph.numNodes, 1.0f / graph.numNodes);
    hipMemcpy(d_pr, pr.data(), graph.numNodes * sizeof(float), hipMemcpyHostToDevice);

    float equalProb = 1.0f / graph.numNodes;
    int blockSize = 256;
    int numBlocks = (graph.numNodes + blockSize - 1) / blockSize;

    for (int i = 0; i < iterations; ++i) {
        pageRankKernel<<<numBlocks, blockSize>>>(graph.numNodes, d_edges, d_offsets, d_pr, d_prNext, damping, equalProb);
        std::swap(d_pr, d_prNext);
    }

    hipMemcpy(pr.data(), d_pr, graph.numNodes * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < graph.numNodes; ++i) {
        std::cout << "Node " << i << ": " << pr[i] << std::endl;
    }

    hipFree(d_pr);
    hipFree(d_prNext);
    freeGraphOnGPU(d_edges, d_offsets);
}

int main(){
    int v, e;
    std::cin >> v;
    Graph graph(v);

    std::cin >> e;
    for(int i = 0; i < e; i++){
        int src, dest;
        std::cin >> src >> dest;
        graph.addEdge(src, dest);
    }
    
    pageRank(graph);

    return 0;
}