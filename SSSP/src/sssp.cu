#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include "graph.h"

__global__ void relaxEdges(Edge* edges, int* distances, int E, bool* updated) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < E) {
        int u = edges[i].src;
        int v = edges[i].dest;
        int weight = edges[i].weight;

        printf("Checking edge %d %d\n", u, v);
        if (distances[u] != INT_MAX && distances[u] + weight < distances[v]) {
            printf("Relaxing edge %d %d\n", u, v);
            distances[v] = distances[u] + weight;
            *updated = true;
        }
    }
}

void sssp(Graph& graph, int src) {
    int V = graph.V;
    int E = graph.E;
    int* distances;
    Edge* d_edges;
    bool* d_updated;

    distances = new int[V];
    for (int i = 0; i < V; ++i) distances[i] = INT_MAX;
    distances[src] = 0;

    hipMalloc(&d_edges, E * sizeof(Edge));
    hipMalloc(&d_updated, sizeof(bool));
    hipMemcpy(d_edges, graph.edges.data(), E * sizeof(Edge), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (E + blockSize - 1) / blockSize;

    for (int i = 0; i < V - 1; ++i) {
        bool updated = false;
        hipMemcpy(d_updated, &updated, sizeof(bool), hipMemcpyHostToDevice);
        relaxEdges<<<numBlocks, blockSize>>>(d_edges, distances, E, d_updated);
        hipMemcpy(&updated, d_updated, sizeof(bool), hipMemcpyDeviceToHost);
        if (!updated) break;
    }

    std::cout << "Vertex Distance from Source\n";
    for (int i = 0; i < V; ++i)
        std::cout << i << "\t\t" << distances[i] << "\n";

    hipFree(d_edges);
    hipFree(d_updated);
    delete[] distances;
}

int main() {
    int V, E;
    std::cin>>V>>E;
    Graph graph(V, E);

    for (int i = 0; i < E; ++i) {
        int src, dest, weight;
        std::cin>>src>>dest>>weight;
        graph.addEdge(src, dest, weight);
    }

    sssp(graph, 0);

    return 0;
}