#include "hip/hip_runtime.h"
#include "graph.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void relaxEdges(Edge* edges, int* distances, int E, bool* updated) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < E) {
        int u = edges[i].src;
        int v = edges[i].dest;
        int weight = edges[i].weight;

        if (distances[u] != INT_MAX && distances[u] + weight < distances[v]) {
            distances[v] = distances[u] + weight;
            *updated = true;
        }
    }
}