#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "graph.h"

#define INF 999999

__global__ void bfs_kernel(int *adjList, int *adjListSizes, int *distances, int *frontier, int *newFrontier, int numNodes, bool *flag_continue) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNodes) return;

    if (frontier[idx]) {
        frontier[idx] = 0;
        int start = adjListSizes[idx];
        int end = adjListSizes[idx + 1];
        for (int i = start; i < end; ++i) {
            int neighbor = adjList[i];
            if (distances[neighbor] == INF) {
                distances[neighbor] = distances[idx] + 1;
                newFrontier[neighbor] = 1;
                *flag_continue = true;
            }
        }
    }
}

void bfs(int startNode, const Graph &graph) {
    int numNodes = graph.getNumNodes();
    const auto &adjListHost = graph.getAdjList();

    int totalEdges = 0;
    std::vector<int> adjList;
    std::vector<int> adjListSizes(numNodes + 1, 0);
    for (int i = 0; i < numNodes; ++i) {
        adjListSizes[i] = totalEdges;
        for (int neighbor : adjListHost[i]) {
            adjList.push_back(neighbor);
            ++totalEdges;
        }
    }
    adjListSizes[numNodes] = totalEdges;

    int *d_adjList, *d_adjListSizes, *d_distances, *d_frontier, *d_newFrontier;
    hipMalloc(&d_adjList, adjList.size() * sizeof(int));
    hipMalloc(&d_adjListSizes, adjListSizes.size() * sizeof(int));
    hipMalloc(&d_distances, numNodes * sizeof(int));
    hipMalloc(&d_frontier, numNodes * sizeof(int));
    hipMalloc(&d_newFrontier, numNodes * sizeof(int));

    std::vector<int> distances(numNodes, INF);
    std::vector<int> frontier(numNodes, 0);
    distances[startNode] = 0;
    frontier[startNode] = 1;

    hipMemcpy(d_adjList, adjList.data(), adjList.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjListSizes, adjListSizes.data(), adjListSizes.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_distances, distances.data(), numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_frontier, frontier.data(), numNodes * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numNodes + blockSize - 1) / blockSize;

    bool *h_continue = new bool;
    bool *d_continue;
    hipMalloc(&d_continue, sizeof(bool));
    do {
        *h_continue = false;
        hipMemcpy(d_continue, h_continue, sizeof(bool), hipMemcpyHostToDevice);

        bfs_kernel<<<numBlocks, blockSize>>>(d_adjList, d_adjListSizes, d_distances, d_frontier, d_newFrontier, numNodes, d_continue);
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            std::cerr << "CUDA error in bfs_kernel: " << hipGetErrorString(error) << std::endl;
            break;
        }

        hipMemcpy(h_continue, d_continue, sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(d_frontier, d_newFrontier, numNodes * sizeof(int), hipMemcpyDeviceToDevice);

    } while (*h_continue);

    hipMemcpy(distances.data(), d_distances, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < numNodes; ++i) {
        std::cout << "Distance to node " << i << ": " << distances[i] << std::endl;
    }

    hipFree(d_adjList);
    hipFree(d_adjListSizes);
    hipFree(d_distances);
    hipFree(d_frontier);
    hipFree(d_newFrontier);
    hipFree(d_continue);
    delete h_continue;
}
